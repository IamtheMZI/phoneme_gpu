// INCLUDES
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#include <iostream>
#include <time.h>

// FUNCTION DEFINITIONS
__global__ void nn_diff(float* input,float* weight, float* output, int column_size);

// DEFINES
#define SIZE 8
#define COLUMN_SIZE 4
#define ROW_SIZE  2


#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stdout, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stdout, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


// OTHER FUNCTIONS
__global__ void nn_diff(float* input,float* weight, float* output, int column_size, int size){
	   int i = blockDim.x * blockIdx.x + threadIdx.x;
	   if (i < size){
		   int in_ind = i%column_size;
		   output[i] = (input[in_ind] - weight[i])*(input[in_ind] - weight[i]);
		   printf("%d %f %d:%f %f\n",i, output[i], in_ind, input[in_ind], weight[i]);
		   __syncthreads();
	   }
}

/*__global__ void nn_diff_add(float* output, float* output_add, int column_size, int size){
	   int i = blockDim.x * blockIdx.x + threadIdx.x;
	   if(i <size){
		   for(int p = 0; p < column_size; p++){
			   output_add[i] += output[p+i*column_size];
			   printf("%d %d:%f %f\n",i, p, output[p+i*column_size], output_add[i]);
		   }
		}
}*/

__global__ void nn_diff_add(float* output, float* output_add, int column_size, int size){
	/*extern*/ __shared__ float sdata[SIZE];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	sdata[tid] = output[i];
	__syncthreads();

	for (unsigned int s=1; s<blockDim.x; s*=2){
		if(tid% (2*s) == 0){
			sdata[tid] += sdata[tid+s];
		}
		__syncthreads();
	}

	if (tid==0) output_add[blockIdx.x]=sdata[0];
}
__device__ int nn_find_minimum(float* output_add){
	float min = 9999999;
	int min_loc = -1;
	for (int idx=0; idx < SIZE/COLUMN_SIZE; idx++){
		if(output_add[idx] < min){
			min = output_add[idx];
			min_loc = idx;
		}
	}
	return min_loc;
}
__global__ void nn_weight_update(float* input, float* weight, float learning_rate, float* output_add, int column_size, int size){
		int location = nn_find_minimum(output_add);
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if(i < size){
		   int idx = location*column_size+i;
		   printf("i:%d location:%d column_size:%d weight:%f index:%d input:%f\n", i, location, column_size, idx, weight[idx], input[i]);
		   weight[idx] = weight[idx] + learning_rate * (input[i] - weight[location*column_size+i]);
		   printf("i:%d location:%d column_size:%d weight:%f index:%d input:%f\n", i, location, column_size, idx, weight[idx], input[i]);
	   }
}



// MAIN FUNCTION
int main(){/*
	// VARIABLES
	float* input;
	float* weight;
	float* output;

	// Allocate Variables
	int in_size = 4;
	input  = (float*) malloc(COLUMN_SIZE*sizeof(float));
	weight = (float*) malloc(SIZE*sizeof(float));
	output = (float*) malloc(in_size*sizeof(float));

	for (	int idx=1; idx < COLUMN_SIZE; idx++){
		 input[idx] = rand() % 10;
	}
	for (	int idx=1; idx < SIZE; idx++){
			weight[idx] = rand() % 10;
	}
	for (	int idx=1; idx < in_size; idx++){
			output[idx] = rand() % 10;
	}*/
	float input[4] = {1.0, 1.0, 0.0, 0.0};
	float weight[8] = {0.2,0.6,0.5,0.9,0.8,0.4,0.7,0.3};
	float output[4] = {0.0};
	float output_add[2] = {0.0};
	float learning_rate = 0.6;
	// Reset the GPUs
	hipDeviceReset();

	// GPU Variable Declaration
	float *dev_input,*dev_weight,*dev_output, *dev_output_add;

	// GPU Variable Allocation
	hipMalloc(&dev_input , sizeof(input));
	hipMalloc(&dev_weight, sizeof(weight));
	hipMalloc(&dev_output, sizeof(output));
	hipMalloc(&dev_output_add, sizeof(output_add));
	cudaCheckErrors("cudamalloc fail");

	// Copy CPU Variable to GPU
	hipMemcpy(dev_input,  input , sizeof(input), hipMemcpyHostToDevice);
	hipMemcpy(dev_weight, weight, sizeof(weight), hipMemcpyHostToDevice);
	hipMemcpy(dev_output, output, sizeof(output), hipMemcpyHostToDevice);
	cudaCheckErrors("cuda memcpy fail");

	nn_diff<<< COLUMN_SIZE ,1 >>>(dev_input,dev_weight,dev_output,COLUMN_SIZE,SIZE); // output = (input - weight)^2
	nn_diff_add<<< ROW_SIZE-1,1 >>>(dev_output,dev_output_add,COLUMN_SIZE,ROW_SIZE); //output_add = Addition of all the columns in a row
	nn_weight_update<<< COLUMN_SIZE ,1 >>>(dev_input,dev_weight,learning_rate, dev_output_add, COLUMN_SIZE,COLUMN_SIZE); //output_add = Addition of all the columns in a row
	hipMemcpy(weight, dev_weight ,sizeof(weight), hipMemcpyDeviceToHost);
	cudaCheckErrors("cudamemcpy or cuda kernel fail");

	for(int idx=0; idx < sizeof(weight)/sizeof(weight[0]); idx++){
		if (idx % COLUMN_SIZE == 0)
			printf("\n");
		printf("%f ",weight[idx]);
	}
	printf("\n");
	hipFree(dev_input);
	hipFree(dev_weight);
	hipFree(dev_output);
	hipFree(dev_output_add);
	return 0;
}
